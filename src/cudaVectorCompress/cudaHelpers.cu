#include "hip/hip_runtime.h"
#include "cudaHelpers.cuh"
#include <hip/hip_runtime.h>
#include <cstring>

hipArray_t Create3DArray(int channelCount, uint4 Dimensions, hipChannelFormatKind ChannelFormat)
{
	hipChannelFormatDesc channelDesc;
	channelDesc.f = ChannelFormat;

	if (ChannelFormat == hipChannelFormatKindFloat)
	{
		channelDesc.x = (channelCount > 0) * 32;
		channelDesc.y = (channelCount > 1) * 32;
		channelDesc.z = (channelCount > 2) * 32;
		channelDesc.w = (channelCount > 3) * 32;
	}

	if (ChannelFormat == cudaChannelFormatKindSignedBlockCompressed6H)
	{
		channelDesc.x = (channelCount > 0) * 16;
		channelDesc.y = (channelCount > 1) * 16;
		channelDesc.z = (channelCount > 2) * 16;
		channelDesc.w = (channelCount > 3) * 0;
	}

	if (ChannelFormat == cudaChannelFormatKindUnsignedBlockCompressed6H)
	{
		channelDesc.x = (channelCount > 0) * 16;
		channelDesc.y = (channelCount > 1) * 16;
		channelDesc.z = (channelCount > 2) * 16;
		channelDesc.w = (channelCount > 3) * 0;
	}

	hipArray_t datasetArray;	// Opaque data buffer optimized for texture fetches
	auto size = make_hipExtent(Dimensions.x, Dimensions.y, Dimensions.z);
	cudaSafeCall(hipMalloc3DArray(&datasetArray, &channelDesc, size, hipArrayDefault));

	return datasetArray;
}

void UploadTo3DArray(const float* SrcPtr, hipArray_t DstPtr)
{
	hipChannelFormatDesc channelDesc;
	hipExtent extents;
	uint flags;
	hipArrayGetInfo(&channelDesc, &extents, &flags, DstPtr);

	int channelCount = (channelDesc.x > 0) + (channelDesc.y > 0) + (channelDesc.z > 0) + (channelDesc.w > 0);
	uint3 Dimensions{ (uint)extents.width, (uint)extents.height, (uint)extents.depth };

	hipMemcpy3DParms CopyParams{};
	auto Extents = make_hipExtent(Dimensions.x, Dimensions.y, Dimensions.z);
	auto BytePerVector = channelCount * sizeof(float);

	CopyParams.srcPtr = make_hipPitchedPtr((float*)SrcPtr, (size_t)Dimensions.x * BytePerVector, Dimensions.x, Dimensions.y);
	CopyParams.dstArray = DstPtr;
	CopyParams.extent = Extents;
	CopyParams.kind = hipMemcpyHostToDevice;
	cudaSafeCall(hipMemcpy3D(&CopyParams));
}

hipTextureObject_t Create3DArrayTexture(hipArray_t ArrayPtr)
{
	hipChannelFormatDesc channelDesc;
	hipExtent extents;
	uint flags;
	hipArrayGetInfo(&channelDesc, &extents, &flags, ArrayPtr);
	int channelCount = (channelDesc.x > 0) + (channelDesc.y > 0) + (channelDesc.z > 0) + (channelDesc.w > 0);
	uint3 Dimensions{ (uint)extents.width, (uint)extents.height, (uint)extents.depth };

	// create texture object
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.linear.devPtr = ArrayPtr;
	resDesc.res.linear.desc = channelDesc;
	resDesc.res.linear.sizeInBytes = (size_t)Dimensions.x * (size_t)Dimensions.y * (size_t)Dimensions.z * (size_t)(channelCount) * sizeof(float);
	//resDesc.res.linear.sizeInBytes = MemoryRequirements.size;

	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 0;
	texDesc.addressMode[0] = hipAddressModeClamp;
	texDesc.addressMode[1] = hipAddressModeClamp;
	texDesc.addressMode[2] = hipAddressModeClamp;
	texDesc.filterMode = hipFilterModeLinear;

	hipTextureObject_t tex = 0;
	cudaSafeCall(hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL));

	return tex;
}
